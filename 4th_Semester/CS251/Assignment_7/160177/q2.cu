
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define ERROR(s) printf("%s \n Usage: %s <no. of elements> <random seed>\n", s, argv[0]); exit(-1);

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                        printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                        exit(-1);\
                                    }\
                             }while(0);

__global__ void Xor(char *dev_in, int num_elements, int interval)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int k = tid * interval;

    if(k >= num_elements)
        return;

    if(k + interval >= num_elements)
        interval = num_elements - k;

    unsigned *tmp1 = (unsigned *)(dev_in + k * sizeof(unsigned));
    unsigned *tmp2 = tmp1 + interval - 1;

    if(tmp1 == tmp2)
        return;

    if(tid % 2 == 0)
    {
        *tmp1 = (*tmp1) ^ (*tmp2);
        *tmp2 = 0;
    }
    else
    {
        *tmp2 = (*tmp1) ^ (*tmp2);
        *tmp1 = 0;
    }
}

int main(int argc, char **argv)
{
    int i, seed, interval;
    unsigned *in;
    char *tmp;
    char *dev_in;
    int blocks, threads = 1024;
    unsigned num_elements;

    if(argc != 3)
    {
        ERROR("Invalid number of parameters!");
    }

    num_elements = atoi(argv[1]);
    if(num_elements<=0)
    {
        ERROR("Invalid number of elements!");
    }

    seed = atoi(argv[2]);
    if(seed<=0)
    {
        ERROR("Invalid seed value!");
    }

    tmp = (char *)malloc(num_elements * sizeof(unsigned));
    in = (unsigned *)tmp;

    srand(seed);
    for(i=0; i<num_elements; i++)
    {
        *in = rand();
        in++;
    }

    hipMalloc(&dev_in, num_elements * sizeof(unsigned));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(dev_in, tmp, num_elements * sizeof(unsigned), hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    blocks = (num_elements + 1023) / threads;

    for(interval=1;;interval++)
    {
        if(blocks > 1)
            blocks = ((num_elements >> interval) + 1023) / threads;
        else
            threads = threads >> 1;

        Xor<<<blocks, threads>>>(dev_in, num_elements, 1 << interval);
        CUDA_ERROR_EXIT("kernel invocation");

        hipMemcpy(tmp, dev_in, num_elements * sizeof(unsigned), hipMemcpyDeviceToHost);
        CUDA_ERROR_EXIT("hipMemcpy");
        in = (unsigned *)tmp;

        if(threads <= 1)
            break;
    }

    hipMemcpy(tmp, dev_in, sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(dev_in);

    in = (unsigned *)tmp;
    printf("%u\n", *in);

    free(tmp);
    return 0;
}
